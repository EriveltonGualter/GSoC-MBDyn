/* $Header$ */
/*
 * MBDyn (C) is a multibody analysis code.
 * http://www.mbdyn.org
 *
 * Copyright (C) 1996-2017
 *
 * Pierangelo Masarati  <masarati@aero.polimi.it>
 *
 * Dipartimento di Ingegneria Aerospaziale - Politecnico di Milano
 * via La Masa, 34 - 20156 Milano, Italy
 * http://www.aero.polimi.it
 *
 * Changing this copyright notice is forbidden.
 *
 * This program is free software; you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation (version 2 of the License).
 * 
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program; if not, write to the Free Software
 * Foundation, Inc., 59 Temple Place, Suite 330, Boston, MA  02111-1307  USA
 */
/*
 * Authors:	Pierangelo Masarati <masarati@aero.polimi.it>
 * 		Tingnan Zhang <tingnan1986@gatech.edu>
 */

#include "mbconfig.h"           /* This goes first in every *.c,*.cc file */

#include <cfloat>
#include <vector>

#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>

#include "cudatest.h"

class ViscoElasticCUDATest : public CUDATest {
private:
	thrust::host_vector<double> m_k_host, m_r_host;
	thrust::device_vector<double> m_k_device, m_r_device, m_f_device;

public:
	ViscoElasticCUDATest(const std::vector<double>& k, const std::vector<double>& r);
	virtual ~ViscoElasticCUDATest(void);
	virtual void GetForce(Vec3& F, const Vec3& X, const Vec3& V);
};

ViscoElasticCUDATest::ViscoElasticCUDATest(const std::vector<double>& k,
const std::vector<double>& r)
: m_k_host(k), m_r_host(r),
m_k_device(k), m_r_device(r),
m_f_device(k.size())
{
}

ViscoElasticCUDATest::~ViscoElasticCUDATest(void)
{
}

// Kelvin-Voight functor
struct kv_functor
{
	const double x, v;

	kv_functor(double x, double v) : x(x), v(v) {};

	__host__ __device__
	double operator()(const double& k, const double& r) const { 
		return -(k*x + r*v);
        };
};

void
ViscoElasticCUDATest::GetForce(Vec3& F, const Vec3& X, const Vec3& V)
{
	thrust::transform(m_k_device.begin(), m_k_device.end(),
		m_r_device.begin(), m_f_device.begin(),
		kv_functor(X(1), V(1)));
	F = Vec3(thrust::reduce(m_f_device.begin(), m_f_device.end(),
			0., thrust::plus<double>()),
		0., 0.);
}

extern "C" void *
mbdyn_CUDATest_init(unsigned n, double *pk, double *pr)
{
	std::vector<double> k(n), r(n);
	for (unsigned i = 0; i < n; i++) {
		k[i] = pk[i];
		r[i] = pr[i];
	}

	return new ViscoElasticCUDATest(k, r);
}

